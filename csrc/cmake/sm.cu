
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hip/hip_runtime_api.h>

#if !defined(CHECK_CUDA)
#  define CHECK_CUDA(e)                                      \
do {                                                         \
    hipError_t code = (e);                                  \
    if (code != hipSuccess) {                               \
        fprintf(stderr, "<%s:%d> %s:\n    %s: %s\n",         \
            __FILE__, __LINE__, #e,                          \
            hipGetErrorName(code),                          \
            hipGetErrorString(code));                       \
        fflush(stderr);                                      \
        exit(1);                                             \
    }                                                        \
} while (0)
#endif

int main() {
    int numSMs = 0;
    CHECK_CUDA(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0));
    printf("%d", numSMs);
}